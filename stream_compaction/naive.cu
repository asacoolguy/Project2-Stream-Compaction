#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
		__global__ void kernNaiveScan(int n, int* input, int* output, int interval) {
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index < n) {
				if (index < interval) {
					output[index] = input[index];
				}
				else if (index >= interval) {
					output[index] = input[index - interval] + input[index];
				}
			}
		}

		__global__ void kernMakeExclusive(int n, int* input, int* output) {
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index < n){
				if (index > 0) {
					output[index] = input[index - 1];
				}
				else {
					output[index] = 0;
				}
			}
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			if (n <= 0) return;

			// allocate 2 arrays on global memory
			int* dev_buffer1;
			int* dev_buffer2;
			size_t sizeInBytes = n * sizeof(int);

			hipMalloc((void**)&dev_buffer1, sizeInBytes);
			checkCUDAError("hipMalloc dev_buffer1 failed!");

			hipMalloc((void**)&dev_buffer2, sizeInBytes);
			checkCUDAError("hipMalloc dev_buffer2 failed!");
			
			// copy the data into global memory
			hipMemcpy(dev_buffer1, idata, sizeInBytes, hipMemcpyHostToDevice);
			checkCUDAError("cudaMemcopy from idata to dev_buffer1 failed!");
			hipMemcpy(dev_buffer2, dev_buffer1, sizeInBytes, hipMemcpyDeviceToDevice);
			checkCUDAError("cudaMemcopy from dev_buffer1 to dev_buffer2 failed!");
			
			int blockSize = 256;
			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

			timer().startGpuTimer();
            
			int ceiling = ilog2ceil(n);
			int** input = &dev_buffer1;
			int** output = &dev_buffer2;

			for (int i = 1; i <= ceiling; i++) {
				int interval = 1 << (i - 1);
				kernNaiveScan << < fullBlocksPerGrid, blockSize >> > (n, *input, *output, interval);
				checkCUDAError("kernNaiveScan failed");
				std::swap(input, output);

			}

			// shift the output array to the right by 1
			kernMakeExclusive << < fullBlocksPerGrid, blockSize >> > (n, *input, *output);
			checkCUDAError("kernMakeExclusive failed");

            timer().endGpuTimer();

			// copy data back 
			hipMemcpy(odata, *output, sizeInBytes, hipMemcpyDeviceToHost);
			checkCUDAError("cudaMemcopy from output to odata failed!");

			// free the allocated arrays
			hipFree(dev_buffer1);
			checkCUDAError("hipFree on dev_buffer1 failed");
			hipFree(dev_buffer2);
			checkCUDAError("hipFree on dev_buffer2 failed");
        }
    }
}
