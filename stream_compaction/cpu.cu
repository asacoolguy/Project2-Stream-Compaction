#include <cstdio>
#include "cpu.h"

#include "common.h"

namespace StreamCompaction {
    namespace CPU {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
	        static PerformanceTimer timer;
	        return timer;
        }

        /**
         * CPU scan (prefix sum).
         * For performance analysis, this is supposed to be a simple for loop.
         * (Optional) For better understanding before starting moving to GPU, you can simulate your GPU scan in this function first.
         */
        void scan(int n, int *odata, const int *idata) {
	        timer().startCpuTimer();
            
			if (n > 0) {
				odata[0] = 0;

				for (int i = 1; i < n; i++) {
					odata[i] = idata[i - 1] + odata[i - 1];
				}
			}

	        timer().endCpuTimer();
        }

        /**
         * CPU stream compaction without using the scan function.
         *
         * @returns the number of elements remaining after compaction.
         */
        int compactWithoutScan(int n, int *odata, const int *idata) {
	        timer().startCpuTimer();
            
			int count = 0;

			for (int i = 0; i < n; i++) {
				if (idata[i] != 0) {
					odata[count++] = idata[i];
				}
			}

	        timer().endCpuTimer();
            return count;
        }

        /**
         * CPU stream compaction using scan and scatter, like the parallel version.
         *
         * @returns the number of elements remaining after compaction.
         */
        int compactWithScan(int n, int *odata, const int *idata) {
	        timer().startCpuTimer();
	        
			int count = 0;

			if (n > 0) {
				
				// first scan
				odata[0] = 0;
				for (int i = 1; i < n; i++) {
					odata[i] = (idata[i - 1] != 0 ? 1 : 0) + odata[i - 1];
				}

				// then scatter
				for (int i = 0; i < n; i++) {
					if (idata[i] != 0) {
						odata[odata[i]] = idata[i];
						count++;
					}
				}
			}

	        timer().endCpuTimer();
            return count;
        }

		/**
		* CPU stream compaction using scan and scatter in a single for loop
		*
		* @returns the number of elements remaining after compaction.
		*/
		int compactWithScanOnePass(int n, int *odata, const int *idata) {
			timer().startCpuTimer();

			int count = 0;

			if (n > 0) {
				odata[0] = 0;
				for (int i = 0; i < n; i++) {
					// first scan
					if (i < n - 1) {
						odata[i + 1] = (idata[i] != 0 ? 1 : 0) + odata[i];
					}
					// then scatter
					if (idata[i] != 0) {
						odata[odata[i]] = idata[i];
						count++;
					}
				}
			}

			timer().endCpuTimer();
			return count;
		}
    }
}
