#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

// #define DEBUG 
		int paddedData[1 << 16];

		__global__ void kernUpSweep(int n, int* buffer, int interval) {
			int index = blockIdx.x * blockDim.x + threadIdx.x;
			if (index < n && index % interval == 0) {
				buffer[index + interval - 1] += buffer[index + (interval >> 1) - 1];
			}
		}

		__global__ void kernDownSweep(int n, int* buffer, int interval, int smallInterval) {
			int index = blockIdx.x * blockDim.x + threadIdx.x;
			if (index < n && index % interval == 0) {
				int t = buffer[index + smallInterval - 1];
				buffer[index + smallInterval - 1] = buffer[index + interval - 1];
				buffer[index + interval - 1] += t;
			}
		}

		__global__ void kernEfficientUpSweep(int n, int* buffer, int interval) {
			int index = (blockIdx.x * blockDim.x + threadIdx.x) * interval;
			if (index < n) {
				buffer[index + interval - 1] += buffer[index + (interval >> 1) - 1];
			}
		}

		__global__ void kernEfficientDownSweep(int n, int* buffer, int interval, int smallInterval) {
			int index = (blockIdx.x * blockDim.x + threadIdx.x) * interval;
			if (index < n) {
				int t = buffer[index + smallInterval - 1];
				buffer[index + smallInterval - 1] = buffer[index + interval - 1];
				buffer[index + interval - 1] += t;
			}
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata, bool efficient) {
			// allocate 2 arrays on global memory. one original. one resized.
			int* dev_padded;

			int logn = ilog2ceil(n);
			int paddedSize = 1 << logn;
			size_t originalSizeInBytes = n * sizeof(int);
			size_t paddedSizeInBytes = paddedSize * sizeof(int);

			hipMalloc((void**)&dev_padded, paddedSizeInBytes);
			checkCUDAError("hipMalloc dev_padded failed!");

			// initialize dev_padded to 0 and copy idata into it
			hipMemset(dev_padded, 0, paddedSizeInBytes);
			checkCUDAError("hipMemset failed");
			hipMemcpy(dev_padded, idata, originalSizeInBytes, hipMemcpyHostToDevice);
			checkCUDAError("cudaMemcopy from idata to dev_original failed!");

			timer().startGpuTimer();

#ifdef DEBUG
				printf("before start: [");
				hipMemcpy(paddedData, dev_padded, paddedSizeInBytes, hipMemcpyDeviceToHost);
				for (int i = 0; i < paddedSize; i++) {
					printf("%d, ", paddedData[i]);
				}
				printf("] \n");
#endif

			scanHelper(paddedSize, logn, dev_padded, efficient);

			timer().endGpuTimer();


			// copy padded data back into odata
			hipMemcpy(odata, dev_padded, originalSizeInBytes, hipMemcpyDeviceToHost);
			checkCUDAError("cudaMemcopy from dev_original to odata failed!");

			// free the allocated arrays
			hipFree(dev_padded);
			checkCUDAError("hipFree on dev_padded failed");
        }


		/**
		* Helper function for scan that does the upsweeps and downsweeps
		*/
		void scanHelper(int n, int logn, int* dev_buffer, bool efficient) {
			
			if (efficient) {
				for (int i = 0; i <= logn - 1; i++) {
					int interval = 1 << (i + 1);
					dim3 numBlocks(((n >> (i + 1)) + Common::blockSize + 1) / Common::blockSize);
					kernEfficientUpSweep << < numBlocks, Common::blockSize >> > (n, dev_buffer, interval);
					checkCUDAError("kernEfficientUpSweep failed!");
				}
			}
			else {
				dim3 fullBlocksPerGrid((n + Common::blockSize - 1) / Common::blockSize);

				for (int i = 0; i <= logn - 1; i++) {
					int interval = 1 << (i + 1);
					kernUpSweep << < fullBlocksPerGrid, Common::blockSize >> > (n, dev_buffer, interval);
					checkCUDAError("kernEfficientUpSweep failed!");
				}
			}

#ifdef DEBUG
			printf("after up sweep: [");
			hipMemcpy(paddedData, dev_buffer, n * sizeof(int), hipMemcpyDeviceToHost);
			for (int i = 0; i < n; i++) {
				printf("%d, ", paddedData[i]);
			}
			printf("] \n");
#endif

			// first set the last value to 0
			hipMemset(dev_buffer + n - 1, 0, sizeof(int));
			checkCUDAError("kernSetIndexInData failed!");

#ifdef DEBUG
			printf("after setting last value: [");
			hipMemcpy(paddedData, dev_buffer, n * sizeof(int), hipMemcpyDeviceToHost);
			for (int i = 0; i < n; i++) {
				printf("%d, ", paddedData[i]);
			}
			printf("] \n");
#endif

			// down sweep
			if (efficient) {
				for (int i = logn - 1; i >= 0; i--) {
					int smallInterval = 1 << i;
					int interval = 1 << (i + 1);
					dim3 numBlocks(((n >> (i + 1)) + Common::blockSize + 1) / Common::blockSize);
					kernEfficientDownSweep << < numBlocks, Common::blockSize >> > (n, dev_buffer, interval, smallInterval);
					checkCUDAError("kernEfficientDownSweep failed!");
				}
			}
			else {
				dim3 fullBlocksPerGrid((n + Common::blockSize - 1) / Common::blockSize);

				for (int i = logn - 1; i >= 0; i--) {
					int smallInterval = 1 << i;
					int interval = 1 << (i + 1);
					kernDownSweep << < fullBlocksPerGrid, Common::blockSize >> > (n, dev_buffer, interval, smallInterval);
					checkCUDAError("kernEfficientDownSweep failed!");
				}
			}

#ifdef DEBUG
			printf("after downsweep: [");
			hipMemcpy(paddedData, dev_buffer, n * sizeof(int), hipMemcpyDeviceToHost);
			for (int i = 0; i < n; i++) {
				printf("%d, ", paddedData[i]);
			}
			printf("] \n");
#endif
		}


        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata, bool efficient) {
			if (n <= 0) return -1;

			int count = 0;

			// ----------------------------------------------
			// ---------- allocate global memory ------------
			// ----------------------------------------------

			int *dev_input, *dev_output, *dev_bools, *dev_indices;
			int logn = ilog2ceil(n);
			int paddedSize = 1 << logn;
			size_t originalSizeInBytes = n * sizeof(int);
			size_t paddedSizeInBytes = paddedSize * sizeof(int);
			dim3 fullBlocksPerGrid((paddedSize + Common::blockSize - 1) / Common::blockSize);

			hipMalloc((void**)&dev_input, paddedSizeInBytes);
			checkCUDAError("hipMalloc dev_input failed!");
			hipMalloc((void**)&dev_output, paddedSizeInBytes);
			checkCUDAError("hipMalloc dev_output failed!");
			hipMalloc((void**)&dev_bools, paddedSizeInBytes);
			checkCUDAError("hipMalloc dev_bools failed!");
			hipMalloc((void**)&dev_indices, paddedSizeInBytes);
			checkCUDAError("hipMalloc dev_indices failed!");

			// ----------------------------------------------------
			// ---------- copy data into global memory ------------
			// ----------------------------------------------------

			// set dev_input to 0 then copy idata into it
			hipMemset(dev_input, 0, paddedSizeInBytes);
			checkCUDAError("hipMemset failed");
			hipMemcpy(dev_input, idata, originalSizeInBytes, hipMemcpyHostToDevice);
			checkCUDAError("cudaMemcopy from idata to dev_original failed!");

			// ---------------------------------------
			// ---------- start algorithm ------------
			// ---------------------------------------

			timer().startGpuTimer();
	
			// turn input into boolean array
			Common::kernMapToBoolean << <fullBlocksPerGrid, Common::blockSize >> > (paddedSize, dev_bools, dev_input);
			checkCUDAError("kernMapToBoolean failed!");

			// exclusive scan the boolean array
			hipMemcpy(dev_indices, dev_bools, paddedSizeInBytes, hipMemcpyDeviceToDevice);
			checkCUDAError("cudaMemcopy from dev_bools to dev_indices failed!");
			scanHelper(paddedSize, logn, dev_indices, efficient);

			// scatter 
			Common::kernScatter << <fullBlocksPerGrid, Common::blockSize >> > (paddedSize, dev_output, dev_input, dev_bools, dev_indices);
			checkCUDAError("kernScatter failed!");

			timer().endGpuTimer();


			// -----------------------------------------------------------
			// ---------- read data from global memory ------------
			// -----------------------------------------------------------

			// first, copy dev_bool into odata to get the count of non-zero elements
			hipMemcpy(odata, dev_bools, originalSizeInBytes, hipMemcpyDeviceToHost);
			checkCUDAError("cudaMemcopy from dev_original to odata failed!");
			for (int i = 0; i < n; i++) {
				if (odata[i] != 0) count++;
			}

			// finally, copy dev_output into odata
			hipMemcpy(odata, dev_output, originalSizeInBytes, hipMemcpyDeviceToHost);
			checkCUDAError("cudaMemcopy from dev_original to odata failed!");

			// ------------------------------------------
			// ---------- free global memory ------------
			// ------------------------------------------

			hipFree(dev_input);
			checkCUDAError("hipFree on dev_input failed");
			hipFree(dev_output);
			checkCUDAError("hipFree on dev_output failed");
			hipFree(dev_bools);
			checkCUDAError("hipFree on dev_bools failed");
			hipFree(dev_indices);
			checkCUDAError("hipFree on dev_indices failed");

			return count;
        }
    }
}
