#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

// #define DEBUG 
		int paddedData[1 << 16];

		__global__ void kernEfficientUpSweep(int n, int* buffer, int interval) {
			// TODO: this interval thing is overloading the limit of int and causing it to roll back
			int index = blockIdx.x * blockDim.x + threadIdx.x;
			if (index < n && index % interval == 0) {
				buffer[index + interval - 1] += buffer[index + (interval >> 1) - 1];
			}
		}

		__global__ void kernEfficientDownSweep(int n, int* buffer, int interval, int smallInterval) {
			int index = blockIdx.x * blockDim.x + threadIdx.x;
			if (index < n && index % interval == 0) {
				int t = buffer[index + smallInterval - 1];
				buffer[index + smallInterval - 1] = buffer[index + interval - 1];
				buffer[index + interval - 1] += t;
			}
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
		// TODO: still has that bug where logn > 13 will crash some programs. maybe something wrong with hipMalloc and memCpy?
        void scan(int n, int *odata, const int *idata) {
			// allocate 2 arrays on global memory. one original. one resized.
			int* dev_padded;
			int* dev_original;

			int logn = ilog2ceil(n);
			int paddedSize = 1 << logn;
			size_t originalSizeInBytes = n * sizeof(int);
			size_t paddedSizeInBytes = paddedSize * sizeof(int);

			dim3 fullBlocksPerGrid((paddedSize + Common::blockSize - 1) / Common::blockSize);

			hipMalloc((void**)&dev_original, originalSizeInBytes);
			checkCUDAError("hipMalloc dev_original failed!");
			hipMalloc((void**)&dev_padded, paddedSizeInBytes);
			checkCUDAError("hipMalloc dev_padded failed!");

			// copy input into dev_original, then copy dev_original into dev_padded
			hipMemcpy(dev_original, idata, originalSizeInBytes, hipMemcpyHostToDevice);
			checkCUDAError("cudaMemcopy from idata to dev_original failed!");
			Common::kernCopyArray << <fullBlocksPerGrid, Common::blockSize >> > (n, paddedSize, dev_original, dev_padded);
			checkCUDAError("kernCopyArray from dev_original to dev_padded failed!");

			timer().startGpuTimer();

#ifdef DEBUG
				printf("before start: [");
				hipMemcpy(paddedData, dev_padded, paddedSizeInBytes, hipMemcpyDeviceToHost);
				for (int i = 0; i < paddedSize; i++) {
					printf("%d, ", paddedData[i]);
				}
				printf("] \n");
#endif

			scanHelper(paddedSize, logn, fullBlocksPerGrid, dev_padded);

			timer().endGpuTimer();


			// copy padded data back into original data
			Common::kernCopyArray << <fullBlocksPerGrid, Common::blockSize >> > (paddedSize, n, dev_padded, dev_original);
			checkCUDAError("kernCopyArray from dev_padded to dev_original failed!");

			hipMemcpy(odata, dev_original, originalSizeInBytes, hipMemcpyDeviceToHost);
			checkCUDAError("cudaMemcopy from dev_original to odata failed!");

			// free the allocated arrays
			hipFree(dev_original);
			checkCUDAError("hipFree on dev_original failed");
			hipFree(dev_padded);
			checkCUDAError("hipFree on dev_padded failed");
        }


		// helper function for scan
		void scanHelper(int n, int logn, dim3 fullBlocksPerGrid, int* dev_buffer) {

			for (int i = 0; i <= logn - 1; i++) {
				int interval = 1 << (i + 1);
				kernEfficientUpSweep << < fullBlocksPerGrid, Common::blockSize >> > (n, dev_buffer, interval);
				checkCUDAError("kernEfficientUpSweep failed!");
			}

#ifdef DEBUG
				printf("after up sweep: [");
				hipMemcpy(paddedData, dev_buffer, n * sizeof(int), hipMemcpyDeviceToHost);
				for (int i = 0; i < n; i++) {
					printf("%d, ", paddedData[i]);
				}
				printf("] \n");
#endif

			// first set the last value to 0
			Common::kernSetIndexInData << <1, 1 >> > (n, n - 1, 0, dev_buffer);
			checkCUDAError("kernSetIndexInData failed!");

#ifdef DEBUG
				printf("after setting last value: [");
				hipMemcpy(paddedData, dev_buffer, n * sizeof(int), hipMemcpyDeviceToHost);
				for (int i = 0; i < n; i++) {
					printf("%d, ", paddedData[i]);
				}
				printf("] \n");
#endif

			// down sweep
			for (int i = logn - 1; i >= 0; i--) {
				int smallInterval = 1 << i;
				int interval = 1 << (i + 1);
				kernEfficientDownSweep << < fullBlocksPerGrid, Common::blockSize >> > (n, dev_buffer, interval, smallInterval);
				checkCUDAError("kernEfficientDownSweep failed!");
			}

#ifdef DEBUG
				printf("after downsweep: [");
				hipMemcpy(paddedData, dev_buffer, n * sizeof(int), hipMemcpyDeviceToHost);
				for (int i = 0; i < n; i++) {
					printf("%d, ", paddedData[i]);
				}
				printf("] \n");
#endif
		}


        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
			if (n <= 0) return -1;

			int count = 0;

			// ----------------------------------------------
			// ---------- allocate global memory ------------
			// ----------------------------------------------

			int *dev_original, *dev_input, *dev_output, *dev_bools, *dev_indices;
			int logn = ilog2ceil(n);
			int paddedSize = 1 << logn;
			size_t originalSizeInBytes = n * sizeof(int);
			size_t paddedSizeInBytes = paddedSize * sizeof(int);

			hipMalloc((void**)&dev_original, originalSizeInBytes);
			checkCUDAError("hipMalloc dev_original failed!");
			hipMalloc((void**)&dev_input, paddedSizeInBytes);
			checkCUDAError("hipMalloc dev_input failed!");
			hipMalloc((void**)&dev_output, paddedSizeInBytes);
			checkCUDAError("hipMalloc dev_output failed!");
			hipMalloc((void**)&dev_bools, paddedSizeInBytes);
			checkCUDAError("hipMalloc dev_bools failed!");
			hipMalloc((void**)&dev_indices, paddedSizeInBytes);
			checkCUDAError("hipMalloc dev_indices failed!");

			dim3 fullBlocksPerGrid((paddedSizeInBytes + Common::blockSize - 1) / Common::blockSize);

			// ----------------------------------------------------
			// ---------- copy data into global memory ------------
			// ----------------------------------------------------

			// first copy the input into dev_original
			hipMemcpy(dev_original, idata, originalSizeInBytes, hipMemcpyHostToDevice);
			checkCUDAError("cudaMemcopy from idata to dev_original failed!");

			// then copy the non-padded dev_original data into the padded dev_input
			Common::kernCopyArray << <fullBlocksPerGrid, Common::blockSize >> > (n, paddedSize, dev_original, dev_input);
			checkCUDAError("kernCopyArray from dev_original to dev_original failed!");

			// ---------------------------------------
			// ---------- start algorithm ------------
			// ---------------------------------------

			timer().startGpuTimer();
	
			// turn input into boolean array
			Common::kernMapToBoolean << <fullBlocksPerGrid, Common::blockSize >> > (paddedSize, dev_bools, dev_input);
			checkCUDAError("kernMapToBoolean failed!");

			// exclusive scan the boolean array
			hipMemcpy(dev_indices, dev_bools, paddedSizeInBytes, hipMemcpyDeviceToDevice);
			checkCUDAError("cudaMemcopy from dev_bools to dev_indices failed!");
			scanHelper(paddedSize, logn, fullBlocksPerGrid, dev_indices);

			// scatter 
			Common::kernScatter << <fullBlocksPerGrid, Common::blockSize >> > (paddedSize, dev_output, dev_input, dev_bools, dev_indices);
			checkCUDAError("kernScatter failed!");

			timer().endGpuTimer();


			// -----------------------------------------------------------
			// ---------- read data from global memory ------------
			// -----------------------------------------------------------

			// first, copy dev_bool into dev_original and read it to get the count of non-zero elements. 
			Common::kernCopyArray << <fullBlocksPerGrid, Common::blockSize >> > (paddedSize, n, dev_bools, dev_original);
			checkCUDAError("kernCopyArray from dev_bools to dev_original failed!");
			hipMemcpy(odata, dev_original, originalSizeInBytes, hipMemcpyDeviceToHost);
			checkCUDAError("cudaMemcopy from dev_original to odata failed!");
			for (int i = 0; i < n; i++) {
				if (odata[i] != 0) count++;
			}

			// finally, copy dev_output into dev_original and read it to get the output
			Common::kernCopyArray << <fullBlocksPerGrid, Common::blockSize >> > (paddedSize, n, dev_output, dev_original);
			checkCUDAError("kernCopyArray from dev_output to dev_original failed!");
			hipMemcpy(odata, dev_original, originalSizeInBytes, hipMemcpyDeviceToHost);
			checkCUDAError("cudaMemcopy from dev_original to odata failed!");

			// ------------------------------------------
			// ---------- free global memory ------------
			// ------------------------------------------

			hipFree(dev_original);
			checkCUDAError("hipFree on dev_original failed");
			hipFree(dev_input);
			checkCUDAError("hipFree on dev_input failed");
			hipFree(dev_output);
			checkCUDAError("hipFree on dev_output failed");
			hipFree(dev_bools);
			checkCUDAError("hipFree on dev_bools failed");
			hipFree(dev_indices);
			checkCUDAError("hipFree on dev_indices failed");

			return count;
        }
    }
}
