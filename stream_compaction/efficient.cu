#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void kernEfficientUpSweep(int n, int* buffer, int interval) {
			int index = (blockIdx.x * blockDim.x) + threadIdx.x * interval;
			if (index < n) {
				buffer[index + interval - 1] += buffer[index + (interval >> 1) - 1];
			}
		}

		__global__ void kernEfficientDownSweep(int n, int* buffer, int interval) {
			int index = (blockIdx.x * blockDim.x) + threadIdx.x * interval;

			if (index < n) {
				int smallInterval = interval >> 1;
				int t = buffer[index + smallInterval - 1];
				buffer[index + smallInterval - 1] = buffer[index + interval - 1];
				buffer[index + interval - 1] += t;
			}
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
		// TODO: for somereason this fails for 2^n where n >= 9. weird. am i doing memcpy wrong?
        void scan(int n, int *odata, const int *idata) {
			// allocate buffer array on global memory
			int* dev_original;
			int* dev_buffer;

			int size = 1 << (ilog2ceil(n));
			size_t originalSizeInBytes = n * sizeof(int);
			size_t sizeInBytes = size * sizeof(int);
			dim3 fullBlocksPerGrid((size + Common::blockSize - 1) / Common::blockSize);

			hipMalloc((void**)&dev_original, originalSizeInBytes);
			checkCUDAError("hipMalloc dev_original failed!");
			hipMalloc((void**)&dev_buffer, sizeInBytes);
			checkCUDAError("hipMalloc dev_buffer failed!");

			// copy the data into global memory
			hipMemcpy(dev_original, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("cudaMemcopy from idata to dev_original failed!");

			// copy the original data into the buffer
			Common::kernCopyArray << <fullBlocksPerGrid, Common::blockSize >> >(n, size, dev_original, dev_buffer);
			checkCUDAError("kernCopy from dev_original to dev_buffer failed!");

			timer().startGpuTimer();

			//printf("before start: [");
			//hipMemcpy(odata, dev_buffer, sizeInBytes, hipMemcpyDeviceToHost);
			//for (int i = 0; i < size; i++) {
			//	printf("%d, ", odata[i]);
			//}
			//printf("] \n");

			scanHelper(n, size, dev_buffer);

			timer().endGpuTimer();

			// copy the buffer data back into the original
			Common::kernCopyArray << <fullBlocksPerGrid, Common::blockSize >> >(size, n, dev_buffer, dev_original);
			checkCUDAError("kernCopy from dev_buffer to dev_original failed!");

			// copy data back 
			hipMemcpy(odata, dev_original, originalSizeInBytes, hipMemcpyDeviceToHost);
			checkCUDAError("cudaMemcopy from dev_original to odata failed!");

			// free the allocated arrays
			hipFree(dev_original);
			checkCUDAError("hipFree on dev_original failed");
			hipFree(dev_buffer);
			checkCUDAError("hipFree on dev_buffer failed");
        }


		// helper function for scan
		void scanHelper(int n, int size, int* dev_buffer) {
			dim3 fullBlocksPerGrid((size + Common::blockSize - 1) / Common::blockSize);

			for (int i = 0; i <= ilog2ceil(n) - 1; i++) {
				int interval = 1 << (i + 1);
				kernEfficientUpSweep << < fullBlocksPerGrid, Common::blockSize >> > (size, dev_buffer, interval);
				checkCUDAError("kernEfficientUpSweep failed!");
			}

			// first set the last value to 0
			Common::kernSetIndexInData << <1, 1 >> > (size, size - 1, 0, dev_buffer);
			checkCUDAError("kernSetIndexInData failed!");

			// down sweep
			for (int i = ilog2ceil(n) - 1; i >= 0; i--) {
				int interval = 1 << (i + 1);
				kernEfficientDownSweep << < fullBlocksPerGrid, Common::blockSize >> > (size, dev_buffer, interval);
				checkCUDAError("kernEfficientDownSweep failed!");
			}
		}


        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
			if (n <= 0) return -1;

			int count = 0;

			// ----------------------------------------------
			// ---------- allocate global memory ------------
			// ----------------------------------------------

			int* dev_input;
			int* dev_output;
			int* dev_bools;
			int* dev_indices;
			int size = 1 << (ilog2ceil(n));
			size_t originalSizeInBytes = n * sizeof(int);
			size_t sizeInBytes = size * sizeof(int);

			hipMalloc((void**)&dev_input, sizeInBytes);
			checkCUDAError("hipMalloc dev_input failed!");
			hipMalloc((void**)&dev_output, sizeInBytes);
			checkCUDAError("hipMalloc dev_output failed!");
			hipMalloc((void**)&dev_bools, sizeInBytes);
			checkCUDAError("hipMalloc dev_bools failed!");
			hipMalloc((void**)&dev_indices, sizeInBytes);
			checkCUDAError("hipMalloc dev_indices failed!");

			hipDeviceSynchronize();

			dim3 fullBlocksPerGrid((size + Common::blockSize - 1) / Common::blockSize);

			// ----------------------------------------------------
			// ---------- copy data into global memory ------------
			// ----------------------------------------------------

			// copy the input into global memory
			hipMemcpy(dev_input, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("cudaMemcopy from idata to dev_original failed!");

			// ---------------------------------------
			// ---------- start algorithm ------------
			// ---------------------------------------

			timer().startGpuTimer();
	
			// turn input into boolean array
			Common::kernMapToBoolean << <fullBlocksPerGrid, Common::blockSize >> > (size, dev_bools, dev_input);
			checkCUDAError("kernMapToBoolean failed!");

			// exclusive scan the boolean array
			hipMemcpy(dev_indices, dev_bools, sizeInBytes, hipMemcpyDeviceToDevice);
			checkCUDAError("cudaMemcopy from dev_bools to dev_indices failed!");
			scanHelper(n, size, dev_indices);

			// scatter 
			Common::kernScatter << <fullBlocksPerGrid, Common::blockSize >> > (n, dev_output, dev_input, dev_bools, dev_indices);
			checkCUDAError("kernScatter failed!");

			timer().endGpuTimer();
			// -----------------------------------------
			// ------------- end algorithm -------------
			// -----------------------------------------


			// -----------------------------------------------------------
			// ---------- read global memory into host memory ------------
			// -----------------------------------------------------------

			// first, read dev_bool to get the count of non-zero elements
			hipMemcpy(odata, dev_bools, originalSizeInBytes, hipMemcpyDeviceToHost);
			checkCUDAError("cudaMemcopy from dev_output to odata failed!");
			for (int i = 0; i < n; i++) {
				if (odata[i] != 0) count++;
			}

			// finally, get the values out of the original array
			hipMemcpy(odata, dev_output, originalSizeInBytes, hipMemcpyDeviceToHost);
			checkCUDAError("cudaMemcopy from dev_output to odata failed!");

			// ------------------------------------------
			// ---------- free global memory ------------
			// ------------------------------------------

			hipFree(dev_input);
			checkCUDAError("hipFree on dev_input failed");
			hipFree(dev_output);
			checkCUDAError("hipFree on dev_output failed");
			hipFree(dev_bools);
			checkCUDAError("hipFree on dev_bools failed");
			hipFree(dev_indices);
			checkCUDAError("hipFree on dev_indices failed");

			return count;
        }
    }
}
